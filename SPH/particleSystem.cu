#include "hip/hip_runtime.h"
// implementation of kernels

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "particleSystem.cuh"


__global__ void kernelComputeDensities() {
	return;
}

__global__ void kernelComputeForces() {
	return;
}

__global__ void kernelComputeCollisions() {
	return;
}

extern "C" {
	void cudaInit(int argc, char** argv) {
		int devID;

		devID = findCudaDevice(argc, (const char**)argv);

		if (devID < 0) {
			printf("No CUDA devices found, exiting\n");
			exit(EXIT_SUCCESS);
		}
	}

	void allocateArray(void** devPtr, size_t size) {
		checkCudaErrors(hipMalloc(devPtr, size));
	}

	void freeArray(void* devPtr) {
		checkCudaErrors(hipFree(devPtr));
	}

	void threadSync() {
		checkCudaErrors(hipDeviceSynchronize());
	}

	void copyArrayFromDevice(void* host, const void* device, size_t size) {
		checkCudaErrors(hipMemcpy((char*)host, device, size, hipMemcpyDeviceToHost));
	}
	
	void copyArrayToDevice(void* device, const void* host, size_t size) {
		checkCudaErrors(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
	}

	void cudaComputeDensities(Particle* dev_particles) {
		return;
	}

	void cudaComputeForces(Particle* dev_particles) {
		return;
	}

	void cudaParticleCollisions(Particle* dev_particles) {
		return;
	}

}