#include "hip/hip_runtime.h"
// implementation of kernels

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "particleSystem.cuh"


__global__ void kernelComputeDensities() {
	return;
}

__global__ void kernelComputeForces() {
	return;
}

__global__ void kernelComputeCollisions() {
	return;
}

extern "C" {
	void cudaInit(int argc, char** argv) {
		int devID;

		devID = findCudaDevice(argc, (const char**)argv);

		if (devID < 0) {
			printf("No CUDA devices found, exiting\n");
			exit(EXIT_SUCCESS);
		}
	}

	void allocateArray(void** devPtr, int size) {
		checkCudaErrors(hipMalloc(devPtr, size));
	}

	void freeArray(void* devPtr) {
		checkCudaErrors(hipFree(devPtr));
	}

	void threadSync() {
		checkCudaErrors(hipDeviceSynchronize());
	}

	void copyArrayFromDevice(void* host, const void* device, int size) {
		checkCudaErrors(hipMemcpy((char*)host, device, size, hipMemcpyDeviceToHost));
	}
	
	void copyArrayToDevice(void* device, const void* host, int size) {
		checkCudaErrors(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
	}

	void cudaComputeDensities() {
		return;
	}

	void cudaComputeForces() {
		return;
	}

	void particleCollisions() {
		return;
	}

}